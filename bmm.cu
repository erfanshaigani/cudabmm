#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"
#include <stdlib.h>
#include <stdio.h>

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
// you may define other macros here!
#define a 4
#if TILEX >= TILEY
	#define K TILEX / TILEY
	#define D (a * TILEX)
	#define BSLoop (a * K)
	#define ASLoop (a)
#else
	#define K TILEY / TILEX	
	#define D (a * TILEY)
	#define BSLoop (a)
	#define ASLoop (a * K)
#endif	
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	// write your GPU kernel function here
	__shared__ float as[TILEY][D];
	__shared__ float bs[D][TILEX];

	int i = TILEY * by + ty;
	int j = TILEX * bx + tx;
	//printf("%d",D);
	int s = 0;
	for(int p = 0; p < n / D; p++)
	{
		//as[ty][tx] = ad[i][p * TILEX + tx];
		//as[ty][tx] = ad[i * n + p * TILEX + tx];

		for(int q = 0; q < ASLoop; q++)
		{
			//as[ty][q * TILEX + tx] = ad[i][p * D + q * TILEX + tx];
			as[ty][q * TILEX + tx] = ad[i * n + p * D + q * TILEX + tx];
		}	

		for(int q = 0; q < BSLoop; q++)
		{
			//bs[q * TILEY + ty][tx] = bd[p * D + q * TILEY + ty][j];
			bs[q * TILEY + ty][tx] = bd[(p * D + q * TILEY + ty) * n + j];
		}
		__syncthreads();
		for (int k = 0; k < D; k++)
		{
			s += as[ty][k] * bs[k][tx];
		}
		__syncthreads();
	}
	//cd[i][j] = s;
	cd[i * n + j] = s;
}
